#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

__global__ void sum_array(int *sum, int *array, int blks, int len) {
    uint32_t s = 0;
    uint32_t start = (len / blks) * blockIdx.x;
    uint32_t end = start + (len / blks);

    for (int i = start; i < end; i++) {
        s += array[i];
    }

    sum[blockIdx.x] = s; 
}

int main() {
    int al = 5e6, nb = 999, i;
    int *array, *result;
    int *d_array, *d_result;
    hipError_t err;

    result = (int *) malloc(sizeof(int) * (1 + nb * al));
    array = result + nb;

    if (result == NULL) {
        printf("Failed to malloc data\n");
        return 1;
    }

    for (i = 0; i < al; i++) {
        array[i] = 16 + i;
    }

    if ((err = hipMalloc((void **) &d_result, sizeof(int) * (1 + nb * al))) != hipSuccess) {
        printf("Failed to malloc data: %s\n", hipGetErrorString(err));
        return 1;
    }
    // cudaMalloc((void **) &d_array, sizeof(int) * nb * al);
    d_array = d_result + nb;

    err = hipMemcpy(d_array, array, al * sizeof(int), hipMemcpyHostToDevice);

    if (err != hipSuccess) {
        printf("err: %s\n", hipGetErrorString(err));
        return 1;
    }

    sum_array<<<nb, 1>>>(d_result, d_array, nb, al);
    hipDeviceSynchronize();
    hipMemcpy(result, d_result, nb * sizeof(int), hipMemcpyDeviceToHost);

    int sum = 0;

    for (i = 0; i < nb; i++) {
        sum += result[i];
    }

    printf("Sum: %d\n", sum);
}

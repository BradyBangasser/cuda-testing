#include <stdio.h>
#include <hip/hip_runtime.h>


__global__ void cuda_hello() {
    printf("Hello World\n");
}

__global__ void cuda_add(int *a, int *b, int *c) {
    *c += *a + *b;
    printf("%d %d %ul\n", threadIdx.x, blockIdx.x, (uint64_t) c);
}

int main() {
    int a = 7, b = 10, c;
    // cuda_hello<<<1024, 1024>>>();

    int *d_a, *d_b, *d_c;

    hipMalloc((void **) &d_a, 3 * sizeof(int));
    d_b = d_a + 1;
    d_c = d_a + 2;

    hipMemcpy(d_a, &a, sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_b, &b, sizeof(int), hipMemcpyHostToDevice);

    cuda_add<<<3, 64>>>(d_a, d_b, d_c);

    hipMemcpy(&c, d_c, sizeof(int), hipMemcpyDeviceToHost);
    hipDeviceSynchronize();
    printf("%d\n", c);
    return 0;
}
